#include "hip/hip_runtime.h"
#include <chrono>
#include <functional>
#include <memory>
#include <string>
#include <Eigen/Dense>
#include "hip/hip_runtime.h"
#include "rclcpp/rclcpp.hpp"
#include <pcl/point_types.h>
#include <pcl_conversions/pcl_conversions.h>
#include <pcl/io/pcd_io.h>

#include "std_msgs/msg/string.hpp"
#include "std_msgs/msg/header.hpp"
#include "sensor_msgs/msg/camera_info.hpp"
#include "sensor_msgs/msg/image.hpp"
#include "sensor_msgs/msg/point_cloud2.hpp"
#include "sensor_msgs/msg/point_field.hpp"
#include <math.h>

#include "cv_bridge/cv_bridge.h"
#define BLOCKS 64
#define imin(a,b) (a<b?a:b)

using namespace std::chrono_literals;
using namespace std;
using namespace Eigen;
using std::placeholders::_1;
template<typename T>
__global__ void dev_matmul(const T *a, const T *b, T *output, int rows){
	//a is 3x3 matrix
	//b is 3x1 set of matrices
	//output is 3x1 set of matrices
	int thread_id= threadIdx.x;
	int block_id = blockIdx.x;
	int offset = block_id*(rows+BLOCKS-1)/BLOCKS + thread_id;

	if(offset < rows){
 		#pragma unroll
		for(int i=0; i < 3; ++i){
			double temp=0;
			for(int k=0; k < 3; ++k){
				temp += a[i*3+k]*b[offset*3 + k];
			}
			output[offset*3 + i] = temp;
		}
	}
}

void matmul(double *a, double *b, double *c){
	//a is 3x3
	//b is 3x1
	#pragma unroll
	for(int i=0; i < 3; ++i){
		double temp=0;
		for(int k=0; k < 3; ++k){
			temp += a[i*3 + k]*b[k]; 
		}
		c[i] = temp;
	}
}

__global__ void dot(double* a, double* b, double* c, int rows) {
	int thread_id = threadIdx.x;
	int block_id = blockIdx.x;

	int offset = block_id*(rows+BLOCKS-1)/BLOCKS + thread_id;
	if(offset < rows){
		double temp=0;
 		#pragma unroll
		for(int i=0; i < 3; ++i){
		    temp += a[i]*b[offset*3+i];    
		}
		c[offset] = temp;
	}
	//if(offset == 0){
		//for(int i=0;i < 3; ++i){
			//printf("gpu : %f ", c[0]);
		//}
		//printf("\n");
	//}
}


void log(hipError_t &&error, int line=0){
	//std::cout << hipGetErrorString(error) << "line : " << line << '\n' << std::flush;
}

class IPM : public rclcpp::Node
{
  public:
    IPM()
    : Node("ipm")
    {
       subscription_caminfo = this->create_subscription<sensor_msgs::msg::CameraInfo>("/camera_forward/camera_info", 10, std::bind(&IPM::call, this, _1));
       subscription_img = this->create_subscription<sensor_msgs::msg::Image>("/camera_forward/image_raw", 10, std::bind(&IPM::process_img, this, _1));
       publisher_ = this->create_publisher<sensor_msgs::msg::PointCloud2>("/igvc/ipm", 10);
    }

  private:
    void call(const sensor_msgs::msg::CameraInfo::SharedPtr msg)
    {
        this->camera_info = *msg;
    }
    void process_img(const sensor_msgs::msg::Image::SharedPtr msg)
    {
	//processing recieved image
	sensor_msgs::msg::PointCloud2 pub_pointcloud;
	unique_ptr<PointCloud> cloud_msg  = std::make_unique<PointCloud>();
	cv_bridge::CvImagePtr cv_ptr = cv_bridge::toCvCopy(msg, sensor_msgs::image_encodings::RGB8);

	cv::Mat gray_image;
	cv::cvtColor(cv_ptr->image, gray_image, cv::COLOR_RGB2GRAY);

	cv::inRange(gray_image, cv::Scalar(245), cv::Scalar(255), gray_image); 
	cv::Mat nonZeroCoordinates;
	cv::findNonZero(gray_image, nonZeroCoordinates);


	//some calculations
	float roll = 0;
	float pitch = 0;//-17 * M_PI / 180;
	float yaw = 0;
	float h = 0.8;
	int m = 3;
	int n = 3;
	vector<double> k(9), nor(3), uv(3);

	double cy, cr, sy, sr, sp, cp;
	cy = cos(yaw);
	sy = sin(yaw);
	cp = cos(pitch);
	sp = sin(pitch);
	cr = cos(roll);
	sr = sin(roll);
	k[0] = cr*cy+sp*sr+sy;
	k[1] = cr*sp*sy-cy*sr;
	k[2] = -cp*sy;
	k[3] = cp*sr;
	k[4] = cp*cr;
	k[5] = sp;
	k[6] = cr*sy-cy*sp*sr;
	k[7] = -cr*cy*sp -sr*sy;
	k[8] = cp*cy;

	nor[0] = 0;
	nor[1] = 1.0;
	nor[2] = 0;

	//what does this do?
	matmul(k.data(), nor.data(), uv.data());

	// no of points to map
	cv::Size s = nonZeroCoordinates.size();
	int rows = s.height;
	std::cout << "rows : " << rows << '\n';
	auto caminfo = this->camera_info.k;
	Eigen::Map<Matrix<double,3,3,RowMajor> > mat(caminfo.data());
	mat = mat.inverse();
	//std::cout << mat;
	double *inv_caminfo = mat.data();

	//for(int i=0;i < 9; ++i){
		//std::cout << inv_caminfo[i] << ' ';
	//}
	//std::cout << '\n';
	//std::cout << mat;
	vector<double> kin_uv(3*rows), uv_hom(3*rows), denom(rows);


	//device
	double *d_uv_hom, *d_kin_uv, *d_caminfo, *d_denom, *d_uv;
	log(hipMalloc((void **) &d_uv_hom, sizeof(double)*3*rows));
	log(hipMalloc((void **) &d_kin_uv, sizeof(double)*3*rows));
	log(hipMalloc((void **) &d_caminfo, sizeof(double)*9));
	log(hipMalloc((void **) &d_denom, sizeof(double)*rows));
	log(hipMalloc((void **) &d_uv, sizeof(double)*3));
 
	 
	//copying to device
	log(hipMemcpy(d_caminfo, inv_caminfo, sizeof(double)*9, hipMemcpyHostToDevice));
	hipMemcpy(d_uv_hom, uv_hom.data(), sizeof(double)*3*rows, hipMemcpyHostToDevice);
	hipMemcpy(d_uv, uv.data(), sizeof(double)*3, hipMemcpyHostToDevice);

	//batch multiplication
	//launching rows no of threads and one block
	//std::cout << "cpu : " << caminfo[8] << '\n';
	dev_matmul<<<BLOCKS, (rows+BLOCKS-1)/BLOCKS>>>(d_caminfo, d_uv_hom, d_kin_uv, rows);
	dot<<<BLOCKS, (rows+BLOCKS-1)/BLOCKS>>>(d_uv, d_kin_uv, d_denom, rows);
	
	hipMemcpy(kin_uv.data(), d_kin_uv, sizeof(double)*3*rows, hipMemcpyDeviceToHost);
	hipMemcpy(denom.data(), d_denom, sizeof(double)*rows, hipMemcpyDeviceToHost);
	

	for(int i=0; i < rows; ++i){
		pcl::PointXYZ vec;
		//fix, make it work im not doing it
		vec.x = h * kin_uv[i*3+2] / denom[i];
		vec.y =  -h * kin_uv[i*3] / denom[i];
		vec.z =  0;//h * kin_uv[i*3+1] / denom[i];
		//std::cout << "value : " << denom[i] << ' ';
		cloud_msg->points.push_back(vec);
	}
	//std::cout << std::endl;

	 hipFree(d_uv_hom);
	 hipFree(d_uv);
	 hipFree(d_kin_uv);
	 hipFree(d_caminfo);
	 hipFree(d_denom);   
	 cloud_msg->height   = 1;
	 cloud_msg->width    = cloud_msg->points.size();
	 cloud_msg->is_dense = false;
	 pcl::toROSMsg(*cloud_msg, pub_pointcloud);
	 pub_pointcloud.header.frame_id = "camera_forward_frame";
	 pub_pointcloud.header.stamp = rclcpp::Clock().now();

	 // Publishing our cloud image
	 publisher_->publish(pub_pointcloud);

	 cloud_msg->points.clear();
    }
    
    rclcpp::Subscription<sensor_msgs::msg::CameraInfo>::SharedPtr subscription_caminfo;
    rclcpp::Subscription<sensor_msgs::msg::Image>::SharedPtr subscription_img;
    rclcpp::Publisher<sensor_msgs::msg::PointCloud2>::SharedPtr publisher_;
    sensor_msgs::msg::CameraInfo camera_info;
    typedef pcl::PointCloud<pcl::PointXYZ> PointCloud;
};



int main(int argc, char * argv[])
{
  rclcpp::init(argc, argv);
  rclcpp::spin(std::make_shared<IPM>());
  rclcpp::shutdown();
  return 0;
}
